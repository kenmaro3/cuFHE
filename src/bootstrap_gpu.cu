#include "hip/hip_runtime.h"
/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <bits/stdint-uintn.h>
#include <include/cufhe.h>
#include <stdio.h>
#include <unistd.h>

#include <include/cufhe_gpu.cuh>
#include <include/bootstrap_gpu.cuh>
#include <include/details/error_gpu.cuh>
#include <include/ntt_gpu/ntt.cuh>
#include <include/encoder.cuh>
#include <iostream>
#include <limits>
#include <vector>

namespace cufhe {
using namespace std;
using namespace TFHEpp;

vector<FFP*> bk_ntts;
vector<CuNTTHandler<>*> ntt_handlers;
vector<lvl0param::T*> ksk_devs;

__global__ void __TRGSW2NTT__(FFP* bk_ntt, TFHEpp::lvl1param::T* bk,
                                          CuNTTHandler<> ntt)
{
    __shared__ FFP sh_temp[lvl1param::n];
    const int index = blockIdx.z * (2 * lvl1param::l * 2 * lvl1param::n) +
                      blockIdx.y * 2 * lvl1param::n + blockIdx.x * lvl1param::n;
    ntt.NTT<lvl1param::T>(&bk_ntt[index], &bk[index], sh_temp, 0);
}

void TRGSW2NTT(cuFHETRGSWNTTlvl1& trgswntt, const TFHEpp::TRGSW<TFHEpp::lvl1param>& trgsw, Stream st){
    hipSetDevice(st.device_id());
    TFHEpp::lvl1param::T* d_trgsw;
    hipMalloc((void**)&d_trgsw, sizeof(trgsw));
    hipMemcpyAsync(d_trgsw, trgsw.data(), sizeof(trgsw), hipMemcpyHostToDevice,st.st());

    dim3 grid(2, 2 * lvl1param::l, 1);
    dim3 block(lvl1param::n >> NTT_THRED_UNITBIT);
    __TRGSW2NTT__<<<grid, block, 0, st.st()>>>(trgswntt.trgswdevices[st.device_id()], d_trgsw,
                                                *ntt_handlers[st.device_id()]);
    CuCheckError();
    hipMemcpyAsync(trgswntt.trgswhost.data(), trgswntt.trgswdevices[st.device_id()],
                    sizeof(trgswntt.trgswhost), hipMemcpyDeviceToHost, st.st());
    hipFree(d_trgsw);
}

void InitializeNTThandlers(const int gpuNum){
    for (int i = 0; i < gpuNum; i++) {
        hipSetDevice(i);

        ntt_handlers.push_back(new CuNTTHandler<>());
        ntt_handlers[i]->Create();
        ntt_handlers[i]->CreateConstant();
        hipDeviceSynchronize();
        CuCheckError();
    }
}

void BootstrappingKeyToNTT(const BootstrappingKey<lvl01param>& bk,
                           const int gpuNum)
{
    bk_ntts.resize(gpuNum);
    for (int i = 0; i < gpuNum; i++) {
        hipSetDevice(i);

        hipMalloc((void**)&bk_ntts[i], sizeof(FFP) * lvl0param::n * 2 *
                                            lvl1param::l * 2 * lvl1param::n);

        TFHEpp::lvl1param::T* d_bk;
        hipMalloc((void**)&d_bk, sizeof(bk));
        hipMemcpy(d_bk, bk.data(), sizeof(bk), hipMemcpyHostToDevice);

        hipDeviceSynchronize();
        CuCheckError();

        dim3 grid(2, 2 * lvl1param::l, lvl0param::n);
        dim3 block(lvl1param::n >> NTT_THRED_UNITBIT);
        __TRGSW2NTT__<<<grid, block>>>(bk_ntts[i], d_bk,
                                                   *ntt_handlers[i]);
        hipDeviceSynchronize();
        CuCheckError();

        hipFree(d_bk);
    }
}

void DeleteBootstrappingKeyNTT(const int gpuNum)
{
    for (int i = 0; i < bk_ntts.size(); i++) {
        hipSetDevice(i);
        hipFree(bk_ntts[i]);

        ntt_handlers[i]->Destroy();
        delete ntt_handlers[i];
    }
    ntt_handlers.clear();
}

void KeySwitchingKeyToDevice(const KeySwitchingKey<lvl10param>& ksk,
                             const int gpuNum)
{
    ksk_devs.resize(gpuNum);
    for (int i = 0; i < gpuNum; i++) {
        hipSetDevice(i);
        hipMalloc((void**)&ksk_devs[i], sizeof(ksk));
        CuSafeCall(hipMemcpy(ksk_devs[i], ksk.data(), sizeof(ksk),
                              hipMemcpyHostToDevice));
    }
}

void DeleteKeySwitchingKey(const int gpuNum)
{
    for (int i = 0; i < ksk_devs.size(); i++) {
        hipSetDevice(i);
        hipFree(ksk_devs[i]);
    }
}

template <class P>
__device__ inline typename P::T modSwitchFromTorus(const uint32_t phase)
{
    constexpr uint32_t Mbit = P::nbit + 1;
    static_assert(32 >= Mbit, "Undefined modSwitchFromTorus!");
    return (phase + (1U << (31 - Mbit))) >> (32 - Mbit);
}

template <class P>
__device__ inline typename P::T modSwitchFromTorusSpecific(const uint32_t phase, int domain_bp)
{
    //return std::round((double)phase/pow(2., domain_bp)*pow(2., target_bp));
    //return (phase >> (domain_bp - target_bp - 1)) % (1UL << (target_bp + 1));
    //return (phase >> (domain_bp - P::targetP::nbit - 1)) % (1UL << (P::targetP::nbit + 1));
    uint32_t tmp = domain_bp - P::nbit - 1;
    //*res = ((*phase + (1U << (tmp-1))) >> tmp) % (1UL<<(P::targetP::nbit + 1)); 
    return ((phase + (1U << (tmp-1))) >> tmp) % (1UL<<(P::nbit + 1));
}


template <class P>
__device__ inline void KeySwitch(typename P::targetP::T* lwe,
                                 const typename P::domainP::T* const tlwe,
                                 const typename P::targetP::T* const ksk)
{
    constexpr typename P::domainP::T decomp_mask = (1U << P::basebit) - 1;
    constexpr typename P::domainP::T decomp_offset =
        1U << (std::numeric_limits<typename P::domainP::T>::digits - 1 -
               P::t * P::basebit);
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();
    for (int i = tid; i <= P::targetP::n; i += bdim) {
        typename P::targetP::T res = 0;
        if (i == P::targetP::n) res = tlwe[P::domainP::n];
        for (int j = 0; j < P::domainP::n; j++) {
            typename P::domainP::T tmp;
            if (j == 0)
                tmp = tlwe[0];
            else
                tmp = -tlwe[P::domainP::n - j];
            tmp += decomp_offset;
            for (int k = 0; k < P::t; k++) {
                typename P::domainP::T val =
                    (tmp >>
                     (std::numeric_limits<typename P::domainP::T>::digits -
                      (k + 1) * P::basebit)) &
                    decomp_mask;
                if (val != 0) {
                    constexpr int numbase = (1 << P::basebit) - 1;
                    res -= ksk[j * (lvl10param::t * numbase *
                                    (P::targetP::n + 1)) +
                               k * (numbase * (P::targetP::n + 1)) +
                               (val - 1) * (P::targetP::n + 1) + i];
                }
            }
        }
        lwe[i] = res;
    }
}

template <class P>
__device__ constexpr typename P::T offsetgen()
{
    typename P::T offset = 0;
    for (int i = 1; i <= P::l; i++)
        offset +=
            P::Bg / 2 *
            (1ULL << (numeric_limits<typename P::T>::digits - i * P::Bgbit));
    return offset;
}

__device__ inline void PolynomialSubAndDecomposition(
    FFP* decpoly, const TFHEpp::lvl1param::T* const poly1, const TFHEpp::lvl1param::T* const poly0)
{
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();
    constexpr uint32_t decomp_mask = (1 << lvl1param::Bgbit) - 1;
    constexpr int32_t decomp_half = 1 << (lvl1param::Bgbit - 1);
    constexpr uint32_t decomp_offset = offsetgen<lvl1param>();
    constexpr typename lvl1param::T roundoffset = 1ULL<<(std::numeric_limits<typename lvl1param::T>::digits-lvl1param::l*lvl1param::Bgbit-1);
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
        // decomp temp
        lvl1param::T temp = poly1[i]-poly0[i]+decomp_offset+roundoffset;
#pragma unroll
        for (int digit = 0; digit < lvl1param::l; digit += 1)
            decpoly[digit * lvl1param::n + i] = FFP(lvl1param::T(
                ((temp >> (std::numeric_limits<typename lvl1param::T>::digits -
                           (digit + 1) * lvl1param::Bgbit)) &
                 decomp_mask) -
                decomp_half));
    }
    __syncthreads();  // must
}

__global__ void __CMUXNTT__(TFHEpp::lvl1param::T* out, const FFP* const tgsw_ntt,
                                  const TFHEpp::lvl1param::T* const tlwe1,
                                  const TFHEpp::lvl1param::T* const tlwe0,
                                  const CuNTTHandler<> ntt)
{
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();

    __shared__ FFP sh[(2 + lvl1param::l + 1) * lvl1param::n];
    FFP* sh_res_ntt = &sh[0];
    FFP* decpoly = &sh[2 * lvl1param::n];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* outtemp =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l) * lvl1param::n];

    PolynomialSubAndDecomposition(decpoly, &tlwe1[0], &tlwe0[0]);

    // l NTTs
    // Input/output/buffer use the same shared memory location.
    if (tid < lvl1param::l * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* tar = &decpoly[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                       << lvl1param::nbit];
        ntt.NTT<FFP>(tar, tar, tar,
                     tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();

// Multiply with bootstrapping key in global memory.
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
        sh_res_ntt[i] = decpoly[0 * lvl1param::n + i] *
                        tgsw_ntt[((2 * 0 + 0) << lvl1param::nbit) + i];
        sh_res_ntt[i + lvl1param::n] =
            decpoly[0 * lvl1param::n + i] *
            tgsw_ntt[((2 * 0 + 1) << lvl1param::nbit) + i];
#pragma unroll
        for (int digit = 1; digit < lvl1param::l; digit += 1) {
            sh_res_ntt[i] += decpoly[digit * lvl1param::n + i] *
                             tgsw_ntt[((2 * digit + 0) << lvl1param::nbit) + i];
            sh_res_ntt[i + lvl1param::n] +=
                decpoly[digit * lvl1param::n + i] *
                tgsw_ntt[((2 * digit + 1) << lvl1param::nbit) + i];
        }
    }
    __syncthreads();

    PolynomialSubAndDecomposition(decpoly, &tlwe1[lvl1param::n], &tlwe0[lvl1param::n]);
    // l NTTs
    // Input/output/buffer use the same shared memory location.
    if (tid < lvl1param::l * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* tar = &decpoly[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                       << lvl1param::nbit];
        ntt.NTT<FFP>(tar, tar, tar,
                     tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();
    // Multiply with bootstrapping key in global memory.
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
#pragma unroll
        for (int digit = 0; digit < lvl1param::l; digit += 1) {
            sh_res_ntt[i] +=
                decpoly[digit * lvl1param::n + i] *
                tgsw_ntt[((2 * (digit + lvl1param::l) + 0) << lvl1param::nbit) +
                         i];
            sh_res_ntt[i + lvl1param::n] +=
                decpoly[digit * lvl1param::n + i] *
                tgsw_ntt[((2 * (digit + lvl1param::l) + 1) << lvl1param::nbit) +
                         i];
        }
    }
    __syncthreads();

    #pragma unroll
    for (int i = tid; i < 2*lvl1param::n; i += bdim) outtemp[i] = tlwe0[i];

    // 2 NTTInvs and add acc
    if (tid < 2 * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* src = &sh_res_ntt[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                          << lvl1param::nbit];
        ntt.NTTInvAdd<typename lvl1param::T>(
            &outtemp[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                             << lvl1param::nbit],
            src, src,
            tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                       << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();  // must
    for(int i = 0; i<2*lvl1param::n;i++) out[i] = outtemp[i];
    __syncthreads();
}

template <class P>
__device__ inline void RotatedTestVector(TFHEpp::lvl1param::T* tlwe,
                                         const int32_t bar,
                                         const typename P::T mu)
{
    // volatile is needed to make register usage of Mux to 128.
    // Reference
    // https://devtalk.nvidia.com/default/topic/466758/cuda-programming-and-performance/tricks-to-fight-register-pressure-or-how-i-got-down-from-29-to-15-registers-/
    volatile const uint32_t tid = ThisThreadRankInBlock();
    volatile const uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i < P::n; i += bdim) {
        tlwe[i] = 0;  // part a
        if (bar == 2 * P::n)
            tlwe[i + P::n] = mu;
        else {
            tlwe[i + P::n] = ((i < (bar & (P::n - 1))) ^ (bar >> P::nbit))
                                 ? -mu
                                 : mu;  // part b
        }
    }
    __syncthreads();
}

template <class P>
__device__ inline void CustomTestVector2(TFHEpp::lvl1param::T* tlwe,
                                         cufhe::EncoderDevice *encoder_domain,
                                         cufhe::EncoderDevice *encoder_target,
                                         double (*function)(double))
{
    // volatile is needed to make register usage of Mux to 128.
    // Reference
    // https://devtalk.nvidia.com/default/topic/466758/cuda-programming-and-performance/tricks-to-fight-register-pressure-or-how-i-got-down-from-29-to-15-registers-/
    volatile const uint32_t tid = ThisThreadRankInBlock();
    volatile const uint32_t bdim = ThisBlockSize();
//#pragma unroll
//    for (int i = tid; i < P::n; i += bdim) {
//        tlwe[i] = 0;  // part a
//        double tmp = encoder_domain->a + encoder_domain->d/2.*double(i)/double(P::n);
//        //tlwe[i + P::n] = encoder_target->encode(function(tmp)); // part b
//        tlwe[i + P::n] = encoder_target->encode(tmp); // part b
//    }
//    __syncthreads();
#pragma unroll
    for (int i = tid; i < P::n; i += bdim) {
        tlwe[i] = 0;  // part a
        //double tmp = encoder_domain->a + encoder_domain->d/2.*double(i)/double(P::n);
        tlwe[i + P::n] = encoder_target->encode(-10);
                             
    }
    __syncthreads();
}

template <class P>
__device__ inline void CustomTestVector(TFHEpp::lvl1param::T* tlwe,
                                         cufhe::EncoderDevice *encoder_domain,
                                         cufhe::EncoderDevice *encoder_target,
                                         double (*function)(double))
{
    // volatile is needed to make register usage of Mux to 128.
    // Reference
    // https://devtalk.nvidia.com/default/topic/466758/cuda-programming-and-performance/tricks-to-fight-register-pressure-or-how-i-got-down-from-29-to-15-registers-/
    volatile const uint32_t tid = ThisThreadRankInBlock();
    volatile const uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i < P::n; i += bdim) {
        tlwe[i] = 0;  // part a
        double tmp = encoder_domain->a + encoder_domain->d/2.*double(i)/double(P::n);
        //tlwe[i + P::n] = encoder_target->encode(function(tmp)); // part b
        tlwe[i + P::n] = encoder_target->encode(tmp); // part b
    }
    __syncthreads();
}

__device__ inline void PolynomialMulByXaiMinusOneAndDecomposition(
    FFP* decpoly, const TFHEpp::lvl1param::T* const poly, const uint32_t a_bar)
{
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();
    constexpr uint32_t decomp_mask = (1 << lvl1param::Bgbit) - 1;
    constexpr int32_t decomp_half = 1 << (lvl1param::Bgbit - 1);
    constexpr uint32_t decomp_offset = offsetgen<lvl1param>();
    constexpr typename lvl1param::T roundoffset = 1ULL<<(std::numeric_limits<typename lvl1param::T>::digits-lvl1param::l*lvl1param::Bgbit-1);
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
        //PolynomialMulByXaiMinus
        lvl1param::T temp = poly[(i - a_bar) & (lvl1param::n - 1)];
        temp = ((i < (a_bar & (lvl1param::n - 1)) ^ (a_bar >> lvl1param::nbit)))
                   ? -temp
                   : temp;
        temp -= poly[i];
        // decomp temp
        temp += decomp_offset + roundoffset;
#pragma unroll
        for (int digit = 0; digit < lvl1param::l; digit += 1)
            decpoly[digit * lvl1param::n + i] = FFP(lvl1param::T(
                ((temp >> (std::numeric_limits<typename lvl1param::T>::digits -
                           (digit + 1) * lvl1param::Bgbit)) &
                 decomp_mask) -
                decomp_half));
    }
    __syncthreads();  // must
}

__device__ inline void Accumulate(TFHEpp::lvl1param::T* tlwe, FFP* sh_res_ntt,
                                  FFP* decpoly, const uint32_t a_bar,
                                  const FFP* const tgsw_ntt,
                                  const CuNTTHandler<> ntt)
{
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();

    PolynomialMulByXaiMinusOneAndDecomposition(decpoly, &tlwe[0], a_bar);

    // l NTTs
    // Input/output/buffer use the same shared memory location.
    if (tid < lvl1param::l * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* tar = &decpoly[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                       << lvl1param::nbit];
        ntt.NTT<FFP>(tar, tar, tar,
                     tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();

// Multiply with bootstrapping key in global memory.
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
        sh_res_ntt[i] = decpoly[0 * lvl1param::n + i] *
                        tgsw_ntt[((2 * 0 + 0) << lvl1param::nbit) + i];
        sh_res_ntt[i + lvl1param::n] =
            decpoly[0 * lvl1param::n + i] *
            tgsw_ntt[((2 * 0 + 1) << lvl1param::nbit) + i];
#pragma unroll
        for (int digit = 1; digit < lvl1param::l; digit += 1) {
            sh_res_ntt[i] += decpoly[digit * lvl1param::n + i] *
                             tgsw_ntt[((2 * digit + 0) << lvl1param::nbit) + i];
            sh_res_ntt[i + lvl1param::n] +=
                decpoly[digit * lvl1param::n + i] *
                tgsw_ntt[((2 * digit + 1) << lvl1param::nbit) + i];
        }
    }
    __syncthreads();

    PolynomialMulByXaiMinusOneAndDecomposition(decpoly, &tlwe[lvl1param::n],
                                               a_bar);
    // l NTTs
    // Input/output/buffer use the same shared memory location.
    if (tid < lvl1param::l * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* tar = &decpoly[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                       << lvl1param::nbit];
        ntt.NTT<FFP>(tar, tar, tar,
                     tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();
    // Multiply with bootstrapping key in global memory.
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
#pragma unroll
        for (int digit = 0; digit < lvl1param::l; digit += 1) {
            sh_res_ntt[i] +=
                decpoly[digit * lvl1param::n + i] *
                tgsw_ntt[((2 * (digit + lvl1param::l) + 0) << lvl1param::nbit) +
                         i];
            sh_res_ntt[i + lvl1param::n] +=
                decpoly[digit * lvl1param::n + i] *
                tgsw_ntt[((2 * (digit + lvl1param::l) + 1) << lvl1param::nbit) +
                         i];
        }
    }
    __syncthreads();

    // 2 NTTInvs and add acc
    if (tid < 2 * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* src = &sh_res_ntt[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                          << lvl1param::nbit];
        ntt.NTTInvAdd<typename lvl1param::T>(
            &tlwe[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                             << lvl1param::nbit],
            src, src,
            tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                       << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();  // must
}

__global__ void __Bootstrap__(TFHEpp::lvl0param::T* out,
                              TFHEpp::lvl0param::T* in,
                              const TFHEpp::lvl1param::T mu,
                              const FFP* const bk,
                              const TFHEpp::lvl0param::T* const ksk,
                              const CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == lvl1param::n);
    __shared__ FFP sh[(2 + lvl1param::l + 1) * lvl1param::n];
    FFP* sh_acc_ntt = &sh[0];
    FFP* decpoly = &sh[2 * lvl1param::n];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l) * lvl1param::n];

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    {
        const uint32_t bar =
            2 * lvl1param::n - modSwitchFromTorus<lvl1param>(in[lvl0param::n]);
        RotatedTestVector<lvl1param>(tlwe, bar, mu);
    }

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // n iterations
        const uint32_t bar = modSwitchFromTorus<lvl1param>(in[i]);
        Accumulate(tlwe, sh_acc_ntt, decpoly, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    KeySwitch<lvl10param>(out, tlwe, ksk);
    __threadfence();
}


__global__ void __ProgrammableBootstrap__(TFHEpp::lvl0param::T* out,
    TFHEpp::lvl0param::T* in,
    cufhe::EncoderDevice *encoder_domain,
    cufhe::EncoderDevice *encoder_target,
    double (*function)(double),
    const FFP* const bk,
    const TFHEpp::lvl0param::T* const ksk,
    const CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == lvl1param::n);
    __shared__ FFP sh[(2 + lvl1param::l + 1) * lvl1param::n];
    FFP* sh_acc_ntt = &sh[0];
    FFP* decpoly = &sh[2 * lvl1param::n];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
    (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l) * lvl1param::n];

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    {
    //const uint32_t a_bar = 2 * lvl1param::n - modSwitchFromTorusSpecific<lvl1param>(in[lvl0param::n], encoder_domain->bp);
    const uint32_t bar = 2 * lvl1param::n - modSwitchFromTorus<lvl1param>(in[lvl0param::n]);
    CustomTestVector2<lvl1param>(tlwe, encoder_domain, encoder_target, function);
    //RotatedTestVector<lvl1param>(tlwe, bar, 1U<<29);
    //AccumuleteInitial(tlwe, sh_acc_ntt, decpoly, bar,
    //           bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);

    ////right now just want to see if this can work
    //TFHEpp::lvl1param::T* poly = &tlwe[0];

    //poly = &tlwe[lvl1param::n];
    //for (int i = 0; i < lvl1param::n; i += 1) {
    //    //lvl1param::T temp = poly[(i - bar) & (lvl1param::n - 1)];
    //    //temp = ((i < (bar & (lvl1param::n - 1)) ^ (bar >> lvl1param::nbit)))
    //    //? -temp
    //    //: temp;
    //    //temp = -temp;
    //    //tlwe[lvl1param::n + i] = temp;
    //    tlwe[lvl1param::n + i] = -tlwe[lvl1param::n + i];
    //    }
    }

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // n iterations
        //const uint32_t bar = modSwitchFromTorusSpecific<lvl1param>(in[i], encoder_domain->bp);
        const uint32_t bar = modSwitchFromTorus<lvl1param>(in[i]);
        Accumulate(tlwe, sh_acc_ntt, decpoly, bar,
        bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    KeySwitch<lvl10param>(out, tlwe, ksk);
    __threadfence();
}



__global__ void __SEandKS__(TFHEpp::lvl0param::T* out, TFHEpp::lvl1param::T* in,
                            FFP* bk, TFHEpp::lvl0param::T* ksk)
{
    KeySwitch<lvl10param>(out, in, ksk);
    __threadfence();
}

__global__ void __BootstrapTLWE2TRLWE__(TFHEpp::lvl1param::T* out,
                                        TFHEpp::lvl0param::T* in,
                                        TFHEpp::lvl1param::T mu, FFP* bk,
                                        CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == lvl1param::n);
    __shared__ FFP sh[(2 + lvl1param::l + 1) * lvl1param::n];
    FFP* sh_acc_ntt = &sh[0];
    FFP* decpoly = &sh[2 * lvl1param::n];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l) * lvl1param::n];

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register uint32_t bar =
        2 * lvl1param::n - modSwitchFromTorus<lvl1param>(in[lvl0param::n]);
    RotatedTestVector<lvl1param>(tlwe, bar, mu);

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // n iterations
        bar = modSwitchFromTorus<lvl1param>(in[i]);
        Accumulate(tlwe, sh_acc_ntt, decpoly, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }
    __syncthreads();
    for (int i = 0; i < 2 * lvl1param::n; i++) {
        out[i] = tlwe[i];
    }
    __threadfence();
}

__global__ void __SEIandBootstrap2TRLWE__(TFHEpp::lvl1param::T* out,
                                        TFHEpp::lvl1param::T* in,
                                        TFHEpp::lvl1param::T mu, FFP* bk,
                                        TFHEpp::lvl0param::T* ksk,
                                        CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == lvl1param::n);
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    FFP* decpoly = &sh[2 * lvl1param::n];

    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l) * lvl1param::n];

    lvl0param::T* tlwelvl0 = (lvl0param::T*)&sh[(2 + lvl1param::l + 1) * lvl1param::n];

    KeySwitch<lvl10param>(tlwelvl0, in, ksk);
    __syncthreads();

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register uint32_t bar =
        2 * lvl1param::n - modSwitchFromTorus<lvl1param>(tlwelvl0[lvl0param::n]);
    RotatedTestVector<lvl1param>(tlwe, bar, mu);

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // n iterations
        bar = modSwitchFromTorus<lvl1param>(tlwelvl0[i]);
        Accumulate(tlwe, sh_acc_ntt, decpoly, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }
    __syncthreads();
    for (int i = 0; i < 2 * lvl1param::n; i++) {
        out[i] = tlwe[i];
    }
    __threadfence();
}

template <int casign, int cbsign, typename lvl0param::T offset>
__device__ inline void __HomGate__(TFHEpp::lvl0param::T* out,
                                   TFHEpp::lvl0param::T* in0,
                                   TFHEpp::lvl0param::T* in1, FFP* bk,
                                   TFHEpp::lvl0param::T* ksk,
                                   CuNTTHandler<> ntt)
{
    __shared__ FFP sh[(2 + lvl1param::l + 1) * lvl1param::n];
    FFP* sh_acc_ntt = &sh[0];
    FFP* decpoly = &sh[2 * lvl1param::n];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l) * lvl1param::n];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    {
        const uint32_t bar =
            2 * lvl1param::n -
            modSwitchFromTorus<lvl1param>(offset + casign * in0[lvl0param::n] +
                                          cbsign * in1[lvl0param::n]);
        RotatedTestVector<lvl1param>(tlwe, bar, lvl1param::mu);
    }

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // lvl0param::n iterations
        const uint32_t bar = modSwitchFromTorus<lvl1param>(0 + casign * in0[i] +
                                                           cbsign * in1[i]);
        Accumulate(tlwe, sh_acc_ntt, decpoly, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }
    KeySwitch<lvl10param>(out, tlwe, ksk);
    __threadfence();
}

__global__ void __NandBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in0,
                                  TFHEpp::lvl0param::T* in1, FFP* bk,
                                  TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<-1, -1, lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __NorBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* in0,
                                 TFHEpp::lvl0param::T* in1, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<-1, -1, -lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __XnorBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in0,
                                  TFHEpp::lvl0param::T* in1, FFP* bk,
                                  TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<-2, -2, -2 * lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __AndBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* in0,
                                 TFHEpp::lvl0param::T* in1, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<1, 1, -lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __OrBootstrap__(TFHEpp::lvl0param::T* out,
                                TFHEpp::lvl0param::T* in0,
                                TFHEpp::lvl0param::T* in1, FFP* bk,
                                TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<1, 1, lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __XorBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* in0,
                                 TFHEpp::lvl0param::T* in1, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<2, 2, 2*lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __AndNYBootstrap__(TFHEpp::lvl0param::T* out,
                                   TFHEpp::lvl0param::T* in0,
                                   TFHEpp::lvl0param::T* in1, FFP* bk,
                                   TFHEpp::lvl0param::T* ksk,
                                   CuNTTHandler<> ntt)
{
    __HomGate__<-1, 1, -lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __AndYNBootstrap__(TFHEpp::lvl0param::T* out,
                                   TFHEpp::lvl0param::T* in0,
                                   TFHEpp::lvl0param::T* in1, FFP* bk,
                                   TFHEpp::lvl0param::T* ksk,
                                   CuNTTHandler<> ntt)
{
    __HomGate__<1, -1, -lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __OrNYBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in0,
                                  TFHEpp::lvl0param::T* in1, FFP* bk,
                                  TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<-1, 1, lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __OrYNBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in0,
                                  TFHEpp::lvl0param::T* in1, FFP* bk,
                                  TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<1, -1, lvl0param::mu>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __CopyBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in)
{
    const uint32_t tid = ThisThreadRankInBlock();
    out[tid] = in[tid];
    __syncthreads();
    __threadfence();
}

__global__ void __NotBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* in)
{
    const uint32_t tid = ThisThreadRankInBlock();
    out[tid] = -in[tid];
    __syncthreads();
    __threadfence();
}

// Mux(inc,in1,in0) = inc?in1:in0 = inc&in1 + (!inc)&in0
__global__ void __MuxBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* inc,
                                 TFHEpp::lvl0param::T* in1,
                                 TFHEpp::lvl0param::T* in0, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    // To use over 48 KiB shared Memory, the dynamic allocation is required.
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    FFP* decpoly = &sh[2 * lvl1param::n];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe1 =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l) * lvl1param::n];
    TFHEpp::lvl1param::T* tlwe0 =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l + 1) * lvl1param::n];
    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register uint32_t bar =
        2 * lvl1param::n -
        modSwitchFromTorus<lvl1param>(-lvl0param::mu + inc[lvl0param::n] +
                                      in1[lvl0param::n]);
    RotatedTestVector<lvl1param>(tlwe1, bar, lvl1param::mu);

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // lvl1param::n iterations
        bar = modSwitchFromTorus<lvl1param>(0 + inc[i] + in1[i]);
        Accumulate(tlwe1, sh_acc_ntt, decpoly, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    bar = 2 * lvl1param::n -
          modSwitchFromTorus<lvl1param>(-lvl0param::mu - inc[lvl0param::n] +
                                        in0[lvl0param::n]);

    RotatedTestVector<lvl1param>(tlwe0, bar, lvl1param::mu);

    for (int i = 0; i < lvl0param::n; i++) {  // lvl1param::n iterations
        bar = modSwitchFromTorus<lvl1param>(0 - inc[i] + in0[i]);
        Accumulate(tlwe0, sh_acc_ntt, decpoly, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    volatile const uint32_t tid = ThisThreadRankInBlock();
    volatile const uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i <= lvl1param::n; i += bdim) {
        tlwe1[i] += tlwe0[i];
        if (i == lvl1param::n) {
            tlwe1[lvl1param::n] += lvl1param::mu;
        }
    }

    __syncthreads();

    KeySwitch<lvl10param>(out, tlwe1, ksk);
    __threadfence();
}

// NMux(inc,in1,in0) = !(inc?in1:in0) = !(inc&in1 + (!inc)&in0)
__global__ void __NMuxBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* inc,
                                 TFHEpp::lvl0param::T* in1,
                                 TFHEpp::lvl0param::T* in0, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    // To use over 48 KiB shared Memory, the dynamic allocation is required.
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    FFP* decpoly = &sh[2 * lvl1param::n];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe1 =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l) * lvl1param::n];
    TFHEpp::lvl1param::T* tlwe0 =
        (TFHEpp::lvl1param::T*)&sh[(2 + lvl1param::l + 1) * lvl1param::n];
    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register uint32_t bar =
        2 * lvl1param::n -
        modSwitchFromTorus<lvl1param>(-lvl0param::mu + inc[lvl0param::n] +
                                      in1[lvl0param::n]);
    RotatedTestVector<lvl1param>(tlwe1, bar, lvl1param::mu);

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // lvl1param::n iterations
        bar = modSwitchFromTorus<lvl1param>(0 + inc[i] + in1[i]);
        Accumulate(tlwe1, sh_acc_ntt, decpoly, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    bar = 2 * lvl1param::n -
          modSwitchFromTorus<lvl1param>(-lvl0param::mu - inc[lvl0param::n] +
                                        in0[lvl0param::n]);

    RotatedTestVector<lvl1param>(tlwe0, bar, lvl1param::mu);

    for (int i = 0; i < lvl0param::n; i++) {  // lvl1param::n iterations
        bar = modSwitchFromTorus<lvl1param>(0 - inc[i] + in0[i]);
        Accumulate(tlwe0, sh_acc_ntt, decpoly, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    volatile const uint32_t tid = ThisThreadRankInBlock();
    volatile const uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i <= lvl1param::n; i += bdim) {
        tlwe1[i] = -tlwe1[i]-tlwe0[i];
        if (i == lvl1param::n) {
            tlwe1[lvl1param::n] -= lvl1param::mu;
        }
    }

    __syncthreads();

    KeySwitch<lvl10param>(out, tlwe1, ksk);
    __threadfence();
}

void Bootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in,
               lvl1param::T mu, hipStream_t st, const int gpuNum)
{
    __Bootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                  st>>>
        (out, in, mu, bk_ntts[gpuNum], ksk_devs[gpuNum], *ntt_handlers[gpuNum]);
    CuCheckError();
}

void ProgrammableBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in,
    hipStream_t st, const int gpuNum, EncoderDevice *encoder_domain, EncoderDevice *encoder_target, double (*function)(double))
{
    __ProgrammableBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
        st>>>
    (out, in, encoder_domain, encoder_target, function, bk_ntts[gpuNum], ksk_devs[gpuNum], *ntt_handlers[gpuNum]);
    CuCheckError();
}


void SEandKS(TFHEpp::lvl0param::T* out, TFHEpp::lvl1param::T* in,
             hipStream_t st, const int gpuNum)
{
    __SEandKS__<<<1, lvl0param::n + 1, 0, st>>>(out, in, bk_ntts[gpuNum],
                                                ksk_devs[gpuNum]);
    CuCheckError();
}

void CMUXNTTkernel(TFHEpp::lvl1param::T* res, const FFP* const cs, TFHEpp::lvl1param::T* const c1, TFHEpp::lvl1param::T* const c0,
                         hipStream_t st, const int gpuNum)
{
    __CMUXNTT__<<<1, lvl1param::l * lvl1param::n>>
                                NTT_THRED_UNITBIT,
                            0, st>>>
        (res, cs, c1, c0, *ntt_handlers[gpuNum]);
    CuCheckError();
}

void BootstrapTLWE2TRLWE(TFHEpp::lvl1param::T* out, TFHEpp::lvl0param::T* in,
                         lvl1param::T mu, hipStream_t st, const int gpuNum)
{
    __BootstrapTLWE2TRLWE__<<<1, lvl1param::l * lvl1param::n>>
                                NTT_THRED_UNITBIT,
                            0, st>>>
        (out, in, mu, bk_ntts[gpuNum], *ntt_handlers[gpuNum]);
    CuCheckError();
}

void SEIandBootstrap2TRLWE(TFHEpp::lvl1param::T* out, TFHEpp::lvl1param::T* in,
                         lvl1param::T mu, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __SEIandBootstrap2TRLWE__), hipFuncAttributeMaxDynamicSharedMemorySize,
        ((2 + lvl1param::l + 1) * lvl1param::n + (lvl0param::n + 1)/2+1) * sizeof(FFP));
    __SEIandBootstrap2TRLWE__<<<1, lvl1param::l * lvl1param::n>>
                                NTT_THRED_UNITBIT,
                            ((2 + lvl1param::l + 1) * lvl1param::n + (lvl0param::n + 1)/2+1) * sizeof(FFP), st>>>
        (out, in, mu, bk_ntts[gpuNum], ksk_devs[gpuNum], *ntt_handlers[gpuNum]);
    CuCheckError();
}

void NandBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                   TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __NandBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                      st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void OrBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                 TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __OrBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                    st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void OrYNBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                   TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __OrYNBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                      st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void OrNYBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                   TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __OrNYBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                      st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void AndBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                  TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __AndBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                     st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void AndYNBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                    TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __AndYNBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                       st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void AndNYBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                    TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __AndNYBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                       st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void NorBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                  TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __NorBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                     st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void XorBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                  TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __XorBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                     st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void XnorBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                   TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    __XnorBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT, 0,
                      st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void CopyBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in,
                   hipStream_t st, const int gpuNum)
{
    __CopyBootstrap__<<<1, lvl0param::n + 1, 0, st>>>(out, in);
    CuCheckError();
}

void NotBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in,
                  hipStream_t st, const int gpuNum)
{
    __NotBootstrap__<<<1, lvl0param::n + 1, 0, st>>>(out, in);
    CuCheckError();
}

void MuxBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* inc,
                  TFHEpp::lvl0param::T* in1, TFHEpp::lvl0param::T* in0,
                  hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __MuxBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        (2 + lvl1param::l + 1 + 1) * lvl1param::n * sizeof(FFP));
    __MuxBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT,
                     (2 + lvl1param::l + 1 + 1) * lvl1param::n * sizeof(FFP),
                     st>>>
        (out, inc, in1, in0, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void NMuxBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* inc,
                  TFHEpp::lvl0param::T* in1, TFHEpp::lvl0param::T* in0,
                  hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __NMuxBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        (2 + lvl1param::l + 1 + 1) * lvl1param::n * sizeof(FFP));
    __NMuxBootstrap__<<<1, lvl1param::l * lvl1param::n>> NTT_THRED_UNITBIT,
                     (2 + lvl1param::l + 1 + 1) * lvl1param::n * sizeof(FFP),
                     st>>>
        (out, inc, in1, in0, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}
}  // namespace cufhe
