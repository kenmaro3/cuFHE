/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <unistd.h>

#include <array>
#include <include/bootstrap_gpu.cuh>
#include <include/cufhe_gpu.cuh>

#include "../thirdparties/TFHEpp/include/cloudkey.hpp"
#include "../thirdparties/TFHEpp/include/params.hpp"

namespace cufhe {

int _gpuNum = 1;

int streamCount = 0;

void SetGPUNum(int gpuNum) { _gpuNum = gpuNum; }

void Initialize()
{
    InitializeNTThandlers(_gpuNum);
}

void Initialize(const TFHEpp::GateKeywoFFT& gk)
{
    InitializeNTThandlers(_gpuNum);
    BootstrappingKeyToNTT(gk.bklvl01, _gpuNum);
    KeySwitchingKeyToDevice(gk.ksk, _gpuNum);
}

void CleanUp()
{
    DeleteBootstrappingKeyNTT(_gpuNum);
    DeleteKeySwitchingKey(_gpuNum);
}

inline void CtxtCopyH2D(Ctxt& c, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(c.tlwedevices[st.device_id()], c.tlwehost.data(),
                    sizeof(c.tlwehost), hipMemcpyHostToDevice, st.st());
}

inline void CtxtCopyD2H(Ctxt& c, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(c.tlwehost.data(), c.tlwedevices[st.device_id()],
                    sizeof(c.tlwehost), hipMemcpyDeviceToHost, st.st());
}

void TRLWElvl1CopyH2D(cuFHETRLWElvl1& c, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(c.trlwedevices[st.device_id()], c.trlwehost.data(),
                    sizeof(c.trlwehost), hipMemcpyHostToDevice, st.st());
}

void TRLWElvl1CopyD2H(cuFHETRLWElvl1& c, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(c.trlwehost.data(), c.trlwedevices[st.device_id()],
                    sizeof(c.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void TRGSWNTTlvl1CopyH2D(cuFHETRGSWNTTlvl1& c, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(c.trgswdevices[st.device_id()], c.trgswhost.data(),
                    sizeof(c.trgswhost), hipMemcpyHostToDevice, st.st());
}

void CMUXNTT(cuFHETRLWElvl1& res, cuFHETRGSWNTTlvl1& cs, cuFHETRLWElvl1& c1, cuFHETRLWElvl1& c0,
                                         Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(cs.trgswdevices[st.device_id()], cs.trgswhost.data(),
                    sizeof(cs.trgswhost), hipMemcpyHostToDevice, st.st());
    hipMemcpyAsync(c1.trlwedevices[st.device_id()], c1.trlwehost.data(),
                    sizeof(c1.trlwehost), hipMemcpyHostToDevice, st.st());
    hipMemcpyAsync(c0.trlwedevices[st.device_id()], c0.trlwehost.data(),
                    sizeof(c0.trlwehost), hipMemcpyHostToDevice, st.st());
    CMUXNTTkernel(res.trlwedevices[st.device_id()], cs.trgswdevices[st.device_id()],
                        c1.trlwedevices[st.device_id()], c0.trlwedevices[st.device_id()], st.st(),
                        st.device_id());
    hipMemcpyAsync(res.trlwehost.data(), res.trlwedevices[st.device_id()],
                    sizeof(res.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void gCMUXNTT(cuFHETRLWElvl1& res, cuFHETRGSWNTTlvl1& cs, cuFHETRLWElvl1& c1, cuFHETRLWElvl1& c0, Stream st)
{
    hipSetDevice(st.device_id());
    CMUXNTTkernel(res.trlwedevices[st.device_id()], cs.trgswdevices[st.device_id()],
                  c1.trlwedevices[st.device_id()], c0.trlwedevices[st.device_id()], st.st(),
                  st.device_id());
}

void GateBootstrappingTLWE2TRLWElvl01NTT(cuFHETRLWElvl1& out, Ctxt& in,
                                         Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in, st);
    BootstrapTLWE2TRLWE(out.trlwedevices[st.device_id()],
                        in.tlwedevices[st.device_id()], 1U << 29, st.st(),
                        st.device_id());
    hipMemcpyAsync(out.trlwehost.data(), out.trlwedevices[st.device_id()],
                    sizeof(out.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void gGateBootstrappingTLWE2TRLWElvl01NTT(cuFHETRLWElvl1& out, Ctxt& in,
                                          Stream st)
{
    hipSetDevice(st.device_id());
    BootstrapTLWE2TRLWE(out.trlwedevices[st.device_id()],
                        in.tlwedevices[st.device_id()], 1U << 29, st.st(),
                        st.device_id());
    hipMemcpyAsync(out.trlwehost.data(), out.trlwedevices[st.device_id()],
                    sizeof(out.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void SampleExtractAndKeySwitch(Ctxt& out, const cuFHETRLWElvl1& in, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(in.trlwedevices[st.device_id()], in.trlwehost.data(),
                    sizeof(in.trlwehost), hipMemcpyHostToDevice, st.st());
    SEandKS(out.tlwedevices[st.device_id()], in.trlwedevices[st.device_id()],
            st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gSampleExtractAndKeySwitch(Ctxt& out, const cuFHETRLWElvl1& in, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(in.trlwedevices[st.device_id()], in.trlwehost.data(),
                    sizeof(in.trlwehost), hipMemcpyHostToDevice, st.st());
    SEandKS(out.tlwedevices[st.device_id()], in.trlwedevices[st.device_id()],
            st.st(), st.device_id());
}

void Nand(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    NandBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gNand(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    NandBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Or(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrBootstrap(out.tlwedevices[st.device_id()],
                in0.tlwedevices[st.device_id()],
                in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gOr(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrBootstrap(out.tlwedevices[st.device_id()],
                in0.tlwedevices[st.device_id()],
                in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void OrYN(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrYNBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gOrYN(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrYNBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void OrNY(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrNYBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gOrNY(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrNYBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void And(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gAnd(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void AndYN(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndYNBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gAndYN(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndYNBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void AndNY(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndNYBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gAndNY(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndNYBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Nor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    NorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gNor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    NorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Xor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    XorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gXor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    XorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Xnor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    XnorBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gXnor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    XnorBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Not(Ctxt& out, Ctxt& in, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in, st);
    NotBootstrap(out.tlwedevices[st.device_id()],
                 in.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gNot(Ctxt& out, Ctxt& in, Stream st)
{
    hipSetDevice(st.device_id());
    NotBootstrap(out.tlwedevices[st.device_id()],
                 in.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Copy(Ctxt& out, Ctxt& in, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in, st);
    CopyBootstrap(out.tlwedevices[st.device_id()],
                  in.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gCopy(Ctxt& out, Ctxt& in, Stream st)
{
    hipSetDevice(st.device_id());
    CopyBootstrap(out.tlwedevices[st.device_id()],
                  in.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void CopyOnHost(Ctxt& out, Ctxt& in) { out.tlwehost = in.tlwehost; }

// Mux(inc,in1,in0) = inc?in1:in0 = inc&in1 + (!inc)&in0
void Mux(Ctxt& out, Ctxt& inc, Ctxt& in1, Ctxt& in0, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(inc, st);
    CtxtCopyH2D(in1, st);
    CtxtCopyH2D(in0, st);
    MuxBootstrap(out.tlwedevices[st.device_id()],
                 inc.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gMux(Ctxt& out, Ctxt& inc, Ctxt& in1, Ctxt& in0, Stream st)
{
    hipSetDevice(st.device_id());
    MuxBootstrap(out.tlwedevices[st.device_id()],
                 inc.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()], st.st(), st.device_id());
}

// void SetToGPU(Ctxt& in)
// {
//     hipMemcpy(in.lwe_sample_device_->data(), in.lwe_sample_->data(),
//                in.lwe_sample_->SizeData(), hipMemcpyHostToDevice);
// }

// void GetFromGPU(Ctxt& out)
// {
//     hipMemcpy(out.lwe_sample_->data(), out.lwe_sample_device_->data(),
//                out.lwe_sample_->SizeData(), hipMemcpyDeviceToHost);
// }

bool StreamQuery(Stream st)
{
    hipSetDevice(st.device_id());
    hipError_t res = hipStreamQuery(st.st());
    if (res == hipSuccess) {
        return true;
    }
    else {
        return false;
    }
}
}  // namespace cufhe
