/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <include/cufhe_gpu.cuh>
#include <include/bootstrap_gpu.cuh>

namespace cufhe {

void Initialize(const PubKey& pub_key) {
  BootstrappingKeyToNTT(pub_key.bk_);
  KeySwitchingKeyToDevice(pub_key.ksk_);
}

void CleanUp() {
  DeleteBootstrappingKeyNTT();
  DeleteKeySwitchingKey();
}

inline void CtxtCopyH2D(const Ctxt& c, Stream st) {
  hipMemcpyAsync(c.lwe_sample_device_->data(),
                  c.lwe_sample_->data(),
                  c.lwe_sample_->SizeData(),
                  hipMemcpyHostToDevice,
                  st.st());
}

inline void CtxtCopyD2H(const Ctxt& c, Stream st) {
  hipMemcpyAsync(c.lwe_sample_->data(),
                  c.lwe_sample_device_->data(),
                  c.lwe_sample_->SizeData(),
                  hipMemcpyDeviceToHost,
                  st.st());
}

void Nand(Ctxt& out,
          const Ctxt& in0,
          const Ctxt& in1,
          Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 8);
  CtxtCopyH2D(in0, st);
  CtxtCopyH2D(in1, st);
  NandBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
  CtxtCopyD2H(out, st);
}

void Or(Ctxt& out,
        const Ctxt& in0,
        const Ctxt& in1,
        Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 8);
  CtxtCopyH2D(in0, st);
  CtxtCopyH2D(in1, st);
  OrBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
  CtxtCopyD2H(out, st);
}

void And(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 8);
  CtxtCopyH2D(in0, st);
  CtxtCopyH2D(in1, st);
  AndBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
  CtxtCopyD2H(out, st);
}

void Nor(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 8);
  CtxtCopyH2D(in0, st);
  CtxtCopyH2D(in1, st);
  NorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
  CtxtCopyD2H(out, st);
}

void Xor(Ctxt& out,
         const Ctxt& in0,
         const Ctxt& in1,
         Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(1, 4);
  CtxtCopyH2D(in0, st);
  CtxtCopyH2D(in1, st);
  XorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
  CtxtCopyD2H(out, st);
}

void Xnor(Ctxt& out,
          const Ctxt& in0,
          const Ctxt& in1,
          Stream st) {
  static const Torus mu = ModSwitchToTorus(1, 8);
  static const Torus fix = ModSwitchToTorus(-1, 4);
  CtxtCopyH2D(in0, st);
  CtxtCopyH2D(in1, st);
  XnorBootstrap(out.lwe_sample_device_, in0.lwe_sample_device_,
      in1.lwe_sample_device_, mu, fix, st.st());
  CtxtCopyD2H(out, st);
}

void Not(Ctxt& out,
         const Ctxt& in,
         Stream st) {
  for (int i = 0; i <= in.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = -in.lwe_sample_->data()[i];
}

void Copy(Ctxt& out,
          const Ctxt& in,
          Stream st) {
  for (int i = 0; i <= in.lwe_sample_->n(); i ++)
    out.lwe_sample_->data()[i] = in.lwe_sample_->data()[i];
}

bool StreamQuery(Stream st)
{
    hipError_t res = hipStreamQuery(st.st());
    if (res == hipSuccess) {
        return true;
    }
    else {
        return false;
    }
}

void ConstantZero(Ctxt& out)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    for (int i = 0; i < out.lwe_sample_->n(); i++) {
        out.lwe_sample_->data()[i] = 0;
    }
    out.lwe_sample_->data()[out.lwe_sample_->n()] = -mu;
}

void ConstantOne(Ctxt& out)
{
    static const Torus mu = ModSwitchToTorus(1, 8);
    for (int i = 0; i < out.lwe_sample_->n(); i++) {
        out.lwe_sample_->data()[i] = 0;
    }
    out.lwe_sample_->data()[out.lwe_sample_->n()] = mu;
}
} // namespace cufhe
